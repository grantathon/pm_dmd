#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2013/2014, March 3 - April 4
// ###
// ###
// ### Evgeny Strekalovskiy, Maria Klodt, Jan Stuehmer, Mohamed Souiai
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###


#include "aux.h"
#include <cstdlib>
#include <iostream>
using std::stringstream;
using std::cerr;
using std::cout;
using std::endl;
using std::string;




// parameter processing: template specialization for T=bool
template<>
bool getParam<bool>(std::string param, bool &var, int argc, char **argv)
{
    const char *c_param = param.c_str();
    for(int i=argc-1; i>=1; i--)
    {
        if (argv[i][0]!='-') continue;
        if (strcmp(argv[i]+1, c_param)==0)
        {
            if (!(i+1<argc) || argv[i+1][0]=='-') { var = true; return true; }
            std::stringstream ss;
            ss << argv[i+1];
            ss >> var;
            return (bool)ss;
        }
    }
    return false;
}




// opencv helpers
void convert_layered_to_interleaved(float *aOut, const float *aIn, int w, int h, int nc)
{
    if (nc==1) { memcpy(aOut, aIn, w*h*sizeof(float)); return; }
    size_t nOmega = (size_t)w*h;
    for (int y=0; y<h; y++)
    {
        for (int x=0; x<w; x++)
        {
            for (int c=0; c<nc; c++)
            {
                aOut[(nc-1-c) + nc*(x + (size_t)w*y)] = aIn[x + (size_t)w*y + nOmega*c];
            }
        }
    }
}
void convert_layered_to_mat(cv::Mat &mOut, const float *aIn)
{
    convert_layered_to_interleaved((float*)mOut.data, aIn, mOut.cols, mOut.rows, mOut.channels());
}


void convert_interleaved_to_layered(float *aOut, const float *aIn, int w, int h, int nc)
{
    if (nc==1) { memcpy(aOut, aIn, w*h*sizeof(float)); return; }
    size_t nOmega = (size_t)w*h;
    for (int y=0; y<h; y++)
    {
        for (int x=0; x<w; x++)
        {
            for (int c=0; c<nc; c++)
            {
                aOut[x + (size_t)w*y + nOmega*c] = aIn[(nc-1-c) + nc*(x + (size_t)w*y)];
            }
        }
    }
}
void convert_mat_to_layered(float *aOut, const cv::Mat &mIn)
{
    convert_interleaved_to_layered(aOut, (float*)mIn.data, mIn.cols, mIn.rows, mIn.channels());
}



void showImage(string title, const cv::Mat &mat, int x, int y)
{
    const char *wTitle = title.c_str();
    cv::namedWindow(wTitle, CV_WINDOW_AUTOSIZE);
    cvMoveWindow(wTitle, x, y);
    cv::imshow(wTitle, mat);
}




// adding Gaussian noise
float noise(float sigma)
{
    float x1 = (float)rand()/RAND_MAX;
    float x2 = (float)rand()/RAND_MAX;
    return sigma * sqrtf(-2*log(std::max(x1,0.000001f)))*cosf(2*M_PI*x2);
}
void addNoise(cv::Mat &m, float sigma)
{
    float *data = (float*)m.data;
    int w = m.cols;
    int h = m.rows;
    int nc = m.channels();
    size_t n = (size_t)w*h*nc;
    for(size_t i=0; i<n; i++)
    {
        data[i] += noise(sigma);
    }
}




// cuda error checking
string prev_file = "";
int prev_line = 0;
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        if (prev_line>0) cout << "Previous CUDA call:" << endl << prev_file << ", line " << prev_line << endl;
        exit(1);
    }
    prev_file = file;
    prev_line = line;
}

#include "hip/hip_runtime.h"
// ###
// ###
// ### Depth Map Denoising of Kinect Depth Images
// ### 
// ###
// ### Technical University of Munich
// ###
// ### 
// ### Grant Bartel, grant.bartel@tum.de
// ### Faisal Caeiro, faisal.caeiro@tum.de
// ### Ayman Saleem, ayman.saleem@tum.de
// ###
// ###

// Uncomment to use the live Kinect Camera
//#define KINECT
#include "aux.h"	// Helping functions for CUDA GPU Programming
#include <iostream>	// For standard IO on console
#include "constant.cuh"

#ifdef KINECT
#include "libfreenect_sync.h"	// Free Kinect Lib
#else
#include <fstream>	// For reading raw binary depth file
#endif

using namespace std;

uint16_t *depth = new uint16_t[KINECT_SIZE_X*KINECT_SIZE_Y];
float *fInDepth = new float[KINECT_SIZE_X*KINECT_SIZE_Y];

void normalizeDepth(uint16_t *input, float *output, bool inverse = false)
{
	uint16_t maxValue = 0.0f;

	// Find the maximum value
	for (size_t y = 0; y < KINECT_SIZE_Y; y++)
	{
		for (size_t x = 0; x < KINECT_SIZE_X; x++)
		{
			size_t idx = x + y * KINECT_SIZE_X;
			if (maxValue < input[idx]) maxValue = input[idx];
		}
	}
	// Normalize it to [0,1]
	for (size_t y = 0; y < KINECT_SIZE_Y; y++)
		for (size_t x = 0; x < KINECT_SIZE_X; x++)
		{
			size_t idx = x + y * KINECT_SIZE_X;
			if (isnan(input[idx])) output[idx] = 1.0f;
			else output[idx] = (inverse) ? 1.0f - (float) input[idx] / (float) maxValue : (float) input[idx] / (float) maxValue;
		}
}

__host__ __device__ float DiamondDotProduct(float *p, int w, int h, int x, int y)
{
    size_t offset = (size_t)h*w;
    float pp = p[0];
    float a1 = 0.0f;    float a2 = 0.0f;    float a3 = 0.0f;
    float b1 = 0.0f;    float b2 = 0.0f;
    float c1 = 0.0f;    float c2 = 0.0f;    float c3 = 0.0f;
    float d1 = 0.0f;    float d2 = 0.0f;
                                            float e3 = 0.0f;

    if(x!=0)            { a1 = p[-1]; a2 = p[offset-1];     a3 = p[2*offset-1]; }
    if((x+1)!=w)        { b1 = p[1];  b2 = p[offset+1]; }
    if(y!=0)            { c1 = p[-w]; c2 = p[offset-w];     c3 = p[2*offset-w]; }
    if((y+1)!=h)        { d1 = p[w];  d2 = p[offset+w]; }
    if(y!=0 && x!=0)    {                                   e3  = p[2*offset-w-1]; }

    return  sqrtf(1.0f/3.0f)*( a1 + b1 + c1 + d1 - 4*pp )
          + sqrtf(2.0f/3.0f)*( c2 + d2 - a2 - b2 )
          + sqrtf(8.0f/3.0f)*( pp + e3 - a3 - c3 );
}

__host__ __device__ void DiamondOperator(float *u, float* dd, int w, int h, int x, int y)
{
    size_t offset = (size_t)h*w;
    float uu = u[0];
    float a = 0.0f;
    float b = 0.0f;
    float c = 0.0f;
    float d = 0.0f;
    float e = 0.0f;

    if(x!=0)                    { a = u[-1]; }
    if((x+1)!=w)                { b = u[1]; }
    if(y!=0)                    { c = u[-w]; }
    if((y+1)!=h)                { d = u[w]; }
    if((y+1)!=h && (x+1)!=w)    { e = u[w+1]; }

    dd[0]           = sqrtf(1.0f/3.0f)*( a + b + c + d - 4*uu );
    dd[offset]      = sqrtf(2.0f/3.0f)*( c + d - a - b );
    dd[2*offset]    = sqrtf(8.0f/3.0f)*( uu + e - b - d );
}

__global__ void ComputeImageUpdate(float *v, float *d, float *p, float *u, int w, int h, float tau, float theta)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    if(x<w && y<h)
    {
        size_t idx = x + (size_t)y*w;
        size_t offset = (size_t)h*w;

        u[idx] = v[idx] - theta*DiamondDotProduct(&p[idx], w, h, x, y);
        
        DiamondOperator(&u[idx], &d[idx], w, h, x, y);
        float p1 = p[idx]           + (tau/theta)*d[idx];
        float p2 = p[idx+offset]    + (tau/theta)*d[idx+offset];
        float p3 = p[idx+2*offset]  + (tau/theta)*d[idx+2*offset];
        float maxDenom = fmax(1, sqrtf(powf(p1, 2) + powf(p2, 2) + powf(p3, 2)));
        
        p[idx]          = p1/maxDenom;
        p[idx+offset]   = p2/maxDenom;
        p[idx+2*offset] = p3/maxDenom;
    }
}

int main(int argc, char **argv)
{
	// Before the GPU can process the kernels, call Device Synchronize for devise initialization
	hipDeviceSynchronize(); CUDA_CHECK;

#ifdef KINECT
#else
	// Raw File input is a must
	string rawfile = "";
	bool ret = getParam("i", rawfile, argc, argv);
	if (!ret) cerr << "ERROR; no input raw file specified" << endl;
	if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> -blockX -blockY -blockZ -theta -tau -decay -N" << endl; return 1;}
#endif

	// Default setting for block sizes
	size_t blockX = 32, blockY = 8, blockZ = 1;
	getParam("blockX", blockX, argc, argv);
	getParam("blockY", blockY, argc, argv);
	getParam("blockZ", blockZ, argc, argv);
	cout << "blocksize: " << blockX << "x" << blockY << "x" << blockZ << endl;

	// Default setting for optimization parameter theta
    float theta = 500.0f;
    getParam("theta", theta, argc, argv);
    cout << "theta: " << theta << endl;

	// Default setting for time step
    float tau = 0.005f;
    getParam("tau", tau, argc, argv);
    cout << "tau: " << tau << endl;

	// Default setting for theta decay
    float decay = 0.98f;
    getParam("decay", decay, argc, argv);
    cout << "decay: " << decay << endl;

	// Default setting for total GPU iterations
    int N = 200;
    getParam("N", N, argc, argv);
    cout << "N: " << N << endl;

#ifdef KINECT
	while (cv::waitKey(30) < 0)
	{
		void *data;
		unsigned int timestamp;
        freenect_sync_get_depth((void**)(&data), &timestamp, 0, FREENECT_DEPTH_11BIT);
        depth = (uint16_t*)data;
      
#else
    // Load the raw file (Size must be KINECT_SIZE_X x KINECT_SIZE_Y) i.e. 640x480
	ifstream file_buf(rawfile.c_str(), ios_base::binary);
	file_buf.read((char*) depth, KINECT_SIZE_X*KINECT_SIZE_Y*sizeof(uint16_t));
	file_buf.close();
#endif

	normalizeDepth(depth, fInDepth);

	// Setup input image and save
	cv::Mat mInDepth(KINECT_SIZE_Y,KINECT_SIZE_X,CV_32FC1);
	convert_layered_to_mat(mInDepth, fInDepth);
	showImage("Input Depth Image", mInDepth, 100, 100);
    cv::imwrite("image_input.png",mInDepth*255.f);

    // Setup output image
    float *fOutDepth = new float[(size_t)KINECT_SIZE_Y*KINECT_SIZE_X];
	cv::Mat mOutDepth(KINECT_SIZE_Y,KINECT_SIZE_X,CV_32FC1);
	
    // Start the timer for the GPU process
    Timer timer;
    timer.start();

    // Allocate memory on the GPU and copy data
    float *dU, *dV, *dP, *dD;
    hipMalloc(&dU, (size_t)KINECT_SIZE_Y*KINECT_SIZE_X*sizeof(float)); CUDA_CHECK;
    hipMalloc(&dV, (size_t)KINECT_SIZE_Y*KINECT_SIZE_X*sizeof(float)); CUDA_CHECK;
    hipMalloc(&dP, (size_t)3*KINECT_SIZE_Y*KINECT_SIZE_X*sizeof(float)); CUDA_CHECK;
    hipMalloc(&dD, (size_t)3*KINECT_SIZE_Y*KINECT_SIZE_X*sizeof(float)); CUDA_CHECK;
    hipMemcpy(dU, fInDepth, (size_t)KINECT_SIZE_Y*KINECT_SIZE_X*sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemcpy(dV, dU, (size_t)KINECT_SIZE_Y*KINECT_SIZE_X*sizeof(float), hipMemcpyDeviceToDevice); CUDA_CHECK;
    hipMemset(dP, 0, (size_t)3*KINECT_SIZE_Y*KINECT_SIZE_X*sizeof(float)); CUDA_CHECK;
    hipMemset(dD, 0, (size_t)3*KINECT_SIZE_Y*KINECT_SIZE_X*sizeof(float)); CUDA_CHECK;

    // Init block and grid sizes
    dim3 block = dim3(blockX, blockY, blockZ);
    dim3 grid = dim3((KINECT_SIZE_X+block.x-1)/block.x, (KINECT_SIZE_Y+block.y-1)/block.y, 1);

    // Iterate through main computation
    for(int n=0; n<N; n++)
    {
        theta *= decay;
        ComputeImageUpdate<<<grid, block>>>(dV, dD, dP, dU, KINECT_SIZE_X, KINECT_SIZE_Y, tau, theta);
        hipDeviceSynchronize();
    }

    // Compute final output image
    theta *= decay;
    ComputeImageUpdate<<<grid, block>>>(dV, dD, dP, dU, KINECT_SIZE_X, KINECT_SIZE_Y, tau, theta);
    
    // Copy data back to CPU
    hipMemcpy(fOutDepth, dU, (size_t)KINECT_SIZE_X*KINECT_SIZE_Y*sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    
    // Deallocate memory on the GPU
    hipFree(dU); CUDA_CHECK;
    hipFree(dV); CUDA_CHECK;
    hipFree(dP); CUDA_CHECK;
    hipFree(dD); CUDA_CHECK;

    // Display output image and save
    convert_layered_to_mat(mOutDepth, fOutDepth);
	showImage("Output Depth Image", mOutDepth, 100+KINECT_SIZE_X, 100);
    cv::imwrite("image_output.png",mOutDepth*255.f);

    // End the timer for the GPU process
    timer.end();
    float t = timer.get();  // Time in seconds
    cout << "GPU time: " << t*1000 << " ms" << endl;

#ifdef KINECT
	}
#else
	// wait for key input to quit
	cv::waitKey(0);
#endif

	// free golbal allocated arrays
	delete[] fInDepth;
	delete[] fOutDepth;
	delete[] depth;
	
	// close all opencv windows
	cvDestroyAllWindows();
	return 0;
}

